
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/Threshold.cu"
#else

#include "../common.h"

void THNN_(Threshold_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           accreal threshold_,
           accreal val_,
           THCudaByteTensor *mask,
           bool inplace)
{
  real threshold = ScalarConvert<accreal, real>::to(threshold_);
  real val = ScalarConvert<accreal, real>::to(val_);
  THCUNN_assertSameGPU(state, 2, input, output);

  if (inplace) {
    if (mask) {
      THCudaByteTensor_resizeNd(state, mask, input->nDimension, input->size, NULL);
      THC_pointwiseApply2(state, input, mask,
        ThresholdUpdateOutputMaskIP<real>(threshold, val)
      );
    } else {
      THC_pointwiseApply1(state, input,
        ThresholdUpdateOutputIP<real>(threshold, val)
      );
    }
    THCTensor_(set)(state, output, input);
  } else {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply2(state, output, input,
      ThresholdUpdateOutput<real>(threshold, val)
    );
  }

  THCudaCheck(cudaGetLastError());
}

void THNN_(Threshold_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           accreal threshold_,
           accreal val_,
           THCudaByteTensor *mask,
           bool inplace)
{
  real threshold = ScalarConvert<accreal, real>::to(threshold_);
  real val = ScalarConvert<accreal, real>::to(val_);
  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 3, input, gradInput, gradOutput);

  // TODO: check that either mask or input is given

  if (mask) {
    THCTensor_(resizeNd)(state, gradInput, mask->nDimension, mask->size, NULL);
    THC_pointwiseApply3(state, gradInput, mask, gradOutput,
       ThresholdUpdateGradInputMask<real>()
    );
  } else if (inplace) {
    THC_pointwiseApply2(state, gradOutput, input,
      ThresholdUpdateGradInputIP<real>(threshold)
    );
    THCTensor_(set)(state, gradInput, gradOutput);
  } else {
    THCTensor_(resizeAs)(state, gradInput, input);
    THC_pointwiseApply3(state, gradInput, input, gradOutput,
       ThresholdUpdateGradInput<real>(threshold)
    );
  }

  THCudaCheck(cudaGetLastError());
}

#endif
